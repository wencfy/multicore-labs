#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define RANGE 17.78

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__ void vecGPU(const float *a, const float *b, float *c, int n, int threads);
/**** end of the kernel declaration ***/

int main(int argc, char *argv[]) {

	int n = 0;			 // number of elements in the arrays
	int i;				 // loop index
	float *a, *b, *c;	 // The arrays that will be processed in the host.
	float *temp;		 // array in host used in the sequential code.
	float *ad, *bd, *cd; // The arrays that will be processed in the device.
	clock_t start, end;	 // to meaure the time taken by a specific part of code

	if (argc != 2) {
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
	}

	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);

	// Allocating the arrays in the host

	if (!(a = (float *)malloc(n * sizeof(float)))) {
		printf("Error allocating array a\n");
		exit(1);
	}

	if (!(b = (float *)malloc(n * sizeof(float)))) {
		printf("Error allocating array b\n");
		exit(1);
	}

	if (!(c = (float *)malloc(n * sizeof(float)))) {
		printf("Error allocating array c\n");
		exit(1);
	}

	if (!(temp = (float *)malloc(n * sizeof(float)))) {
		printf("Error allocating array temp\n");
		exit(1);
	}

	// Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n; i++) {
		a[i] = ((float)rand() / (float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand() / (float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand() / (float)(RAND_MAX)) * RANGE;
		temp[i] = c[i]; // temp is just another copy of C
	}

	// The sequential part
	start = clock();
	for (i = 0; i < n; i++)
		temp[i] += a[i] * b[i];
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

	/******************  The start GPU part: Do not modify anything in main() above this line  ************/
	// The GPU part

	/* TODO: in this part you need to do the following:
		1. allocate ad, bd, and cd in the device
		2. send a, b, and c to the device
	*/
	hipMalloc(&ad, n * sizeof(float));
	hipMalloc(&bd, n * sizeof(float));
	hipMalloc(&cd, n * sizeof(float));
	
	start = clock();

	hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bd, b, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cd, c, n * sizeof(float), hipMemcpyHostToDevice);

	/* TODO:
		3. write the kernel, call it: vecGPU
		4. call the kernel (the kernel itself will be written at the comment at the end of this file),
		   you need to decide about the number of threads, blocks, etc and their geometry.
	*/
	dim3 threadsPerBlock(500);
	dim3 numBlocks(8);

	vecGPU<<<threadsPerBlock, numBlocks>>>(ad, bd, cd, n, threadsPerBlock.x * numBlocks.x);

	end = clock();
	/* TODO:
		5. bring the cd array back from the device and store it in c array (declared earlier in main)
		6. free ad, bd, and cd
	*/
	hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(ad);
	hipFree(bd);
	hipFree(cd);

	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/

	// checking the correctness of the GPU part
	for (i = 0; i < n; i++)
		if (fabs(temp[i] - c[i]) >= 0.009) // compare up to the second degit in floating point
			printf("Element %d in the result array does not match the sequential version\n", i);

	// Free the arrays in the host
	free(a);
	free(b);
	free(c);
	free(temp);

	return 0;
}

/**** TODO: Write the kernel itself below this line *****/
__global__ void vecGPU(const float *a, const float *b, float *c, int n, int threads) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = index; i < n; i += threads) {
		c[i] += a[i] * b[i];
	}
}